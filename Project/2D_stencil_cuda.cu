
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>


#define BLOCK_SIZE 32
#define DSIZE 1024
#define RADIUS 3

__global__ void stencil_2D(int *in, int *out){

    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS][BLOCK_SIZE + 2 * RADIUS];
    int gindex_x = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex_x = threadIdx.x + RADIUS;
    int gindex_y = threadIdx.y + blockIdx.y * blockDim.y;
    int lindex_y = threadIdx.y + RADIUS;

    //read input elements into shared memory
    int size = DSIZE + 2 * RADIUS;
    temp[lindex_x][lindex_y] = in[gindex_y + size * gindex_x];

    //Take care of boundary conditions
    if (threadIdx.x < RADIUS){
        temp[lindex_x - RADIUS][lindex_y] = in[gindex_y + size * (gindex_x - RADIUS)];
        temp[lindex_x + BLOCK_SIZE][lindex_y] = in[gindex_y + size * (gindex_x + BLOCK_SIZE)];
    }
    if (threadIdx.y < RADIUS){
        temp[lindex_x][lindex_y - RADIUS] = in[gindex_y - RADIUS + size * gindex_x];
        temp[lindex_x][lindex_y + BLOCK_SIZE] = in[gindex_y + BLOCK_SIZE + size * gindex_x];
    }

    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++){
        result += temp[lindex_x + offset][lindex_y];
        result += temp[lindex_x][lindex_y + offset];
        }
    result -= temp[lindex_x][lindex_y];

    out[gindex_y + size * gindex_x] = result;
}

__global__ void mult_square_matrix(int *a, int *b, int *c, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    // Make sure we are not out of range
    if ((idx < size) && (idy < size)) {
        int temp = 0;
        for (int i = 0; i < size; i++){
            temp += a[idy*size+i] * b[i*size+idx];
        }
        c[idy*size+idx] = temp;                    
    }
}

bool check_matrix_mult(int *a, int *b, int *c, int size){
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            int temp = 0;
            for (int k = 0; k < size; k++){
                temp += a[i*size+k] * b[k*size+j];
            }
            if (temp != c[i*size+j]){
                printf("Error! at %d, %d. Expected %d, got %d", i, j, temp, c[i*size+j]);
                return false;
            }
        }
    }
    return true;
}

bool check_matrix_stencil(int *a, int *b, int size){
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            int temp = 0;
            if (i < RADIUS || i >= size - RADIUS || j < RADIUS || j >= size - RADIUS){
                temp = a[i*size+j];
            }
            else{
                for (int k = -RADIUS; k <= RADIUS; k++){
                    temp += a[(i+k)*size+j];
                    temp += a[i*size+j+k];
                }
                temp -= a[i*size+j];
            }
            if (temp != b[i*size+j]){
                printf("Error! at %d, %d. Expected %d, got %d", i, j, temp, b[i*size+j]);
                return false;
            }
        }
    }
    return true;
}

int main(void){
    int *A, *B, *C, *D, *E;
    int *d_A, *d_B, *d_C, *d_D, *d_E;

    int size = (DSIZE + 2 * RADIUS) * (DSIZE + 2 * RADIUS) * sizeof(int);
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    D = (int *)malloc(size);
    E = (int *)malloc(size);

    //Fill arrays with integers
    for(int i = 0; i < (DSIZE + 2 * RADIUS) * (DSIZE + 2 * RADIUS); i++){
        A[i] = rand() % 10;
        B[i] = rand() % 10;
        C[i] = A[i];
        D[i] = B[i];
        E[i] = 0;
    }

    //Allocate memory on the device for stencil operation
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
    hipMalloc((void **)&d_D, size);
    hipMalloc((void **)&d_E, size);

    // //Copy from host to device for the 2D stencil operation
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, size, hipMemcpyHostToDevice);
    hipMemcpy(d_E, E, size, hipMemcpyHostToDevice);

    //Set up the execution configuration for the 2D stencil operation
    int gridSize = (DSIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(gridSize, gridSize);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    //Launch 2D stencil kernel for both A and B
    stencil_2D<<<dimGrid, dimBlock>>>(d_A + RADIUS + RADIUS * (DSIZE + 2 * RADIUS), d_C + RADIUS + RADIUS * (DSIZE + 2 * RADIUS));
    stencil_2D<<<dimGrid, dimBlock>>>(d_B + RADIUS + RADIUS * (DSIZE + 2 * RADIUS), d_D + RADIUS + RADIUS * (DSIZE + 2 * RADIUS));
    
    //unneeded copy back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);

    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2((DSIZE + 2 * RADIUS), (DSIZE + 2 * RADIUS));
    mult_square_matrix<<<dimGrid2, dimBlock2>>>(d_C, d_D, d_E, DSIZE+2*RADIUS);

    //copy d_E back to host
    hipMemcpy(E, d_E, size, hipMemcpyDeviceToHost);

    //Check if the multiplication was correct
    if(not check_matrix_mult(C,D,E,DSIZE + 2 * RADIUS)){
        printf("Matrix multiplication failed\n");
        exit(1);
    }
    else{
        printf("Matrix multiplication Successful\n");
    }

    if(not check_matrix_stencil(A,C,DSIZE + 2 * RADIUS)){
        printf("Stencil A failed\n");
        exit(1);
    }
    else{
        printf("Stencil A Successful\n");
    }

    if(not check_matrix_stencil(B,D,DSIZE + 2 * RADIUS)){
        printf("Stencil B failed\n");
        exit(1);
    }
    else{
        printf("Stencil B Successful\n");
    }


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);

    //Free host memory
    free(A);
    free(B);
    free(C);
    free(D);
    free(E);

}

